#include "hip/hip_runtime.h"


#include "gpu_transform.h"

// __global__ void matMultKernel(const float* A, const float*B, float* C, int m, int n, int k)

__global__ void GPUTransform(const pcl::PointXYZ *pts, pcl::PointXYZ *cloud_out_pts, const Eigen::Matrix4f *matrix, int n_pts)
{

        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        float sum = 0;  

        if (col < n_pts && row < 4)
        {
                
                printf("hi ");
                // sum += (*matrix)(row, 0) * pts[col].x;
                // sum += (*matrix)(row, 1) * pts[col].y;
                // sum += (*matrix)(row, 2) * pts[col].z;
                // sum += (*matrix)(row, 3); //translation
                // printf("OK! -> npts: %d, r: %d , c: %d. sum: %f\n", n_pts, row,col,sum);
                // cloud_out_pts[col].data[row] = sum;           
        }
        
}

namespace gpu
{

        void TransformUnified(const pcl::PointCloud<pcl::PointXYZ> *cloud, pcl::PointCloud<pcl::PointXYZ> *cloud_out, const Eigen::Matrix4f *transform)
        {

                int n_pts = cloud->size();
                    if(cloud_out->size() != n_pts){
                            printf("resizing ... \n");
                       cloud_out->resize(n_pts);
                    }

                hipError_t err;

                const pcl::PointXYZ *pts = cloud->points.data();

                for(int i =0; i < cloud->size(); ++ i){
                        printf("pt[%d]: %f %f %f\n",i,pts[i].x, pts[i].y, pts[i].z);
                }

                int m = 4;
                int n = 4;
                int k = n_pts;

                int threads_per_block = 4; //
                unsigned int grid_rows = (m + threads_per_block - 1) / threads_per_block;
                unsigned int grid_cols = (k + threads_per_block - 1) / threads_per_block;
                dim3 dimGrid(grid_cols, grid_rows);
                dim3 dimBlock(threads_per_block, threads_per_block);
                printf("dimGrid (%d,%d,%d)\n",dimGrid.x,dimGrid.y,dimGrid.z);
                printf("dimBlock (%d,%d,%d)\n",dimBlock.x,dimBlock.y,dimBlock.z);
                GPUTransform<<<dimGrid, dimBlock>>>(cloud->points.data(), cloud_out->points.data(), transform, n_pts);

                err = hipGetLastError();
                if (err != hipSuccess)
                        std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;

                // err = hipMemcpy(cloud_out->points.data(), d_tf_pts, n_pts * sizeof(pcl::PointXYZ), hipMemcpyDeviceToHost);
                // if (err != hipSuccess)
                //         std::cout << "hipMemcpy cloud out Failure" << std::endl;


        }

        void Transform(const pcl::PointCloud<pcl::PointXYZ> &cloud, pcl::PointCloud<pcl::PointXYZ> &cloud_out, const Eigen::Matrix4f &transform)
        {
                pcl::PointXYZ *d_pts;
                Eigen::Matrix4f *d_transform_matrix;
                pcl::PointXYZ *d_tf_pts;

                int n_pts = cloud.points.size();

                if (cloud_out.points.size() != n_pts)
                        cloud_out.points.resize(n_pts);

                hipError_t err;

                err = hipMalloc(&d_pts, n_pts * sizeof(pcl::PointXYZ));
                if (err != hipSuccess)
                        std::cout << "hipMalloc Failure" << std::endl;

                err = hipMalloc(&d_transform_matrix, sizeof(Eigen::Matrix4f));
                if (err != hipSuccess)
                        std::cout << "hipMalloc Failure" << std::endl;

                err = hipMalloc(&d_tf_pts, n_pts * sizeof(pcl::PointXYZ));
                if (err != hipSuccess)
                        std::cout << "hipMalloc Failure" << std::endl;

                std::cout << "GPU Mem Used: " << (2 * n_pts * sizeof(pcl::PointXYZ) + sizeof(Eigen::Matrix4f)) / 1000000 << " MB" << std::endl;

                err = hipMemcpy(d_pts, cloud.points.data(), n_pts * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                        std::cout << "hipMalloc Failure" << std::endl;

                err = hipMemcpy(d_transform_matrix, &transform, sizeof(Eigen::Matrix4f), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                        std::cout << "hipMalloc Failure" << std::endl;

                // Kernel Call
                int m = 4;
                int n = 4;
                int k = n_pts;

                int threads_per_block = 4; //
                unsigned int grid_rows = (m + threads_per_block - 1) / threads_per_block;
                unsigned int grid_cols = (k + threads_per_block - 1) / threads_per_block;
                dim3 dimGrid(grid_cols, grid_rows);
                dim3 dimBlock(threads_per_block, threads_per_block);
                // printf("dimGrid (%d,%d,%d)\n",dimGrid.x,dimGrid.y,dimGrid.z);
                // printf("dimBlock (%d,%d,%d)\n",dimBlock.x,dimBlock.y,dimBlock.z);
                GPUTransform<<<dimGrid, dimBlock>>>(d_pts, d_tf_pts, d_transform_matrix, n_pts);

                err = hipGetLastError();
                if (err != hipSuccess)
                        std::cout << "Kernel launch error: " << hipGetErrorString(err) << std::endl;

                err = hipMemcpy(cloud_out.points.data(), d_tf_pts, n_pts * sizeof(pcl::PointXYZ), hipMemcpyDeviceToHost);
                if (err != hipSuccess)
                        std::cout << "hipMemcpy Failure" << std::endl;

                hipFree(d_pts);
                hipFree(d_tf_pts);
                hipFree(d_pts);
        }

}
